#include "hip/hip_runtime.h"
// 	Multithreads with syncthreads block optimizer and Update way of cacaulate time
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <iostream>

using namespace std;

#define DATA_SIZE 1048576
#define MY_THREAD_NUM 256
#define BLOCK_NUM 128
//32 * 256 = 81902 threads
int data[DATA_SIZE];

void GenerateNumbers(int *number, int size)
{
    for (int i = 0; i < size; i++) {
        number[i] = i % 10;
    }
}
__global__ static void sumOfSquares(int *num, int* result)
{
    //声明一块共享内存
    extern __shared__ int shared[];
    //表示目前的thread是第几个thread(由0开始计算)
    const int tid = threadIdx.x;
    //表示目前的thread是第几个block(从0开始计算)
    const int bid = blockIdx.x;
    //计算每个线程需要完成的量
//    const int size = DATA_SIZE / MY_THREAD_NUM;
    shared[tid] = 0;
    int sum = 0;
    int i;
    //多线程使用运行内存连续优化技巧
//    for (i = tid; i < DATA_SIZE; i+=MY_THREAD_NUM) {
//        sum += num[i] * num[i] * num[i];
//    }
    //普通多线程
//    for (i = tid * size; i < (tid+1)*size; i++) {
//        sum += num[i] * num[i] * num[i];
//    }
    //多线程使用block和内存连续优化
//    for(i = bid * MY_THREAD_NUM + tid; i < DATA_SIZE; i+=BLOCK_NUM*MY_THREAD_NUM){
//        sum += num[i] * num[i] * num[i];
//    }
    //thread需要同时通过tid和bid来确定，同时不要忘记保证内存连续性
    for (i = bid * MY_THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * MY_THREAD_NUM) {
        shared[tid] += num[i] * num[i] * num[i];
    }
    //同步 保证每个 thread 都已经把结果写到 shared[tid] 里面
    __syncthreads();
    //使用线程0完成加和
    //树装加法
    int offset = 1, mask = 1;
    while(offset < MY_THREAD_NUM){
        if((tid&mask) == 0){
            shared[tid] += shared[tid+offset];
        }
        offset += offset;
        mask += offset;
        __syncthreads();
    }
    if(tid == 0){
        result[bid] = shared[0];
    }
}

int Cal_Squares_Sum(){
    //生成随机数
    GenerateNumbers(data, DATA_SIZE);
    //把数据复制到显卡内存中
    int* gpudata, *result;
    //hipMalloc 取得一块显卡内存 ( 其中result用来存储计算结果 )
    hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
    hipMalloc((void**)&result, sizeof(int)*BLOCK_NUM);
    //hipMemcpy 将产生的随机数复制到显卡内存中
    //hipMemcpyHostToDevice - 从内存复制到显卡内存
    //hipMemcpyDeviceToHost - 从显卡内存复制到内存
    hipMemcpy(gpudata, data, sizeof(int)* DATA_SIZE, hipMemcpyHostToDevice);
    // 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
    //使用event计算时间
    float time_elapsed=0;
    hipEvent_t start,stop;
    hipEventCreate(&start);    //创建Event
    hipEventCreate(&stop);
    hipEventRecord( start,0);    //记录当前时间
    sumOfSquares <<<BLOCK_NUM, MY_THREAD_NUM, MY_THREAD_NUM * sizeof(int) >>>(gpudata, result);
    hipEventRecord( stop,0);    //记录当前时间
    hipEventSynchronize(start);    //Waits for an event to complete.
    hipEventSynchronize(stop);    //Waits for an event to complete.Record之前的任务
    //把结果从显示芯片复制回主内存
    int sum[BLOCK_NUM];
    //hipMemcpy 将结果从显存中复制回内存
    hipMemcpy(&sum, result, sizeof(int) *  BLOCK_NUM, hipMemcpyDeviceToHost);
    //used_time是GPU的时钟周期（timestamp），需要除以GPU的运行频率才能得到以秒为单位的时间
    //Free
    int final_sum = 0;
    for(int i = 0; i < BLOCK_NUM; i++){
        final_sum += sum[i];
    }
    hipEventElapsedTime(&time_elapsed,start,stop);    //计算时间差
    hipFree(gpudata);
    hipFree(result);
    //采取新的计时策略 把每个 block 最早的开始时间，和最晚的结束时间相减，取得总运行时间
    hipEventDestroy(start);    //destory the event
    hipEventDestroy(stop);
    printf("GPU sum: %d GPU time: %.10f ms\n", final_sum, time_elapsed);

    final_sum = 0;
    clock_t cpu_start_time = clock();
    for(int i=0; i<DATA_SIZE; i++){
        final_sum += data[i] * data[i] * data[i];
    }
    clock_t cpu_used_time = clock()-cpu_start_time;
    double cpu_time = (double)(cpu_used_time)/CLOCKS_PER_SEC*1000.0;
    printf("CPU sum: %d CPU time: %.10f ms\n", final_sum, (double)(cpu_used_time)/CLOCKS_PER_SEC*1000.0);
    printf("Speed Ratio %.10f\n", 2.555/time_elapsed);
    return 1;
}