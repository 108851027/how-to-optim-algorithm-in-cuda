#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <algorithm>
#include <iostream>
#include <hip/hip_fp16.h>
using namespace std;

// FastAdd is referenced from
// https://github.com/pytorch/pytorch/blob/396c3b1d88d7624938a2bb0b287f2a19f1e89bb4/aten/src/ATen/native/cuda/KernelUtils.cuh#L29
template<typename T, typename std::enable_if<std::is_same<half, T>::value>::type* = nullptr>
__device__ __forceinline__ void FastSpecializedAtomicAdd(T* base, size_t offset,
                                                         const size_t length, T value) {
#if ((defined(CUDA_VERSION) && (CUDA_VERSION < 10000)) \
     || (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)))
  cuda::atomic::Add(reinterpret_cast<half*>(base) + offset, static_cast<half>(value));
#else
  // Accounts for the chance base falls on an odd 16 bit alignment (ie, not 32 bit aligned)
  __half* target_addr = reinterpret_cast<__half*>(base + offset);
  bool low_byte = (reinterpret_cast<std::uintptr_t>(target_addr) % sizeof(__half2) == 0);

  if (low_byte && offset < (length - 1)) {
    __half2 value2;
    value2.x = value;
    value2.y = __float2half_rz(0);
    cuda::atomic::Add(reinterpret_cast<__half2*>(target_addr), value2);

  } else if (!low_byte && offset > 0) {
    __half2 value2;
    value2.x = __float2half_rz(0);
    value2.y = value;
    cuda::atomic::Add(reinterpret_cast<__half2*>(target_addr - 1), value2);

  } else {
    cuda::atomic::Add(reinterpret_cast<__half*>(base) + offset, static_cast<__half>(value));
  }
#endif
}

template<typename T, typename std::enable_if<!std::is_same<half, T>::value>::type* = nullptr>
__device__ __forceinline__ void FastSpecializedAtomicAdd(T* base, size_t offset,
                                                         const size_t length, T value) {
  cuda::atomic::Add(base + offset, value);
}

template<class T>
__device__ __forceinline__ void FastAdd(T* base, size_t offset, const size_t length, T value) {
  FastSpecializedAtomicAdd(base, offset, length, value);
}

// vector inner product

template<typename T>
__global__ void dot(T* a, T* b, T* c, int n){
    const int nStep = gridDim.x * blockDim.x;
    double temp = 0.0;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    while (gid < n) {
        temp += a[gid] * b[gid];
        gid += nStep;
    }
    atomicAdd(c, temp);
}

int main(){
    
}

