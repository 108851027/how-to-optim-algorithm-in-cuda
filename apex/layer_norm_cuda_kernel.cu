#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/AccumulateType.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/DeviceUtils.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "type_shim.h"
#include "static_switch.h"

// 这段代码定义了一个名为 cuWelfordOnlineSum 的 CUDA 设备函数，它实现了 Welford 算法用于在线计算均值（mu）和方差（sigma2）。
// Welford 算法是一种数值稳定的方法，用于逐步计算一系列数据的均值和方差。
// const U curr: 当前要处理的值。
// U& mu: 到目前为止所有值的均值。
// U& sigma2: 到目前为止所有值的方差。
// U& count: 到目前为止处理的元素数量。
template<typename U> __device__
void cuWelfordOnlineSum(
  const U curr,
  U& mu,
  U& sigma2,
  U& count)
{
  count = count + U(1); // 更新元素计数，每次调用函数时增加 1。
  U delta = curr - mu; // 计算当前值和当前均值的差值。
  U lmean = mu + delta / count; // 根据差值和元素数量计算新的均值。
  mu = lmean; // 将计算出的新均值赋值给 mu。
  U delta2 = curr - lmean; // 计算当前值和新均值的差值。
  sigma2 = sigma2 + delta * delta2; // 根据新旧均值之间的差值更新方差。
}

// 这段代码定义了一个名为 cuChanOnlineSum 的 CUDA 设备函数，它是另一种在线算法，
// 用于更新均值（mu）和方差（sigma2），考虑了两个独立样本的合并。
// const U muB, sigma2B, countB: 分别代表第二组数据的均值、方差和元素数量。
// U& mu, sigma2, count: 代表当前累积（第一组数据）的均值、方差和元素数量，这些将被更新以反映合并后的新值。
template<typename U> __device__
void cuChanOnlineSum(
  const U muB,
  const U sigma2B,
  const U countB,
  U& mu,
  U& sigma2,
  U& count)
{
  U delta = muB - mu; // 计算两组数据均值之间的差。
  U nA = count; // 保存当前组（A组）的元素数量。
  U nB = countB; // 获取第二组（B组）的元素数量。
  count = count + countB; // 更新元素总数。
  U nX = count; // 新的总元素数量。
  if (nX > U(0)) {
    nA = nA / nX; // 计算两组数据在新总数中的相对比例。
    nB = nB / nX; 
    mu = nA*mu + nB*muB; // 根据比例和各自的均值计算新的总均值。
    sigma2 = sigma2 + sigma2B + delta * delta * nA * nB * nX; // 更新方差，考虑两组数据方差和均值差的影响。
  } else {
    // 如果新的总数 nX 为 0，表明两组数据都是空的，因此将 mu 和 sigma2 设置为 0。
    mu = U(0);
    sigma2 = U(0);
  }
}

// 这段代码定义了一个名为 cuRMSOnlineSum 的 CUDA 设备函数，用于在线计算平方和，
// 从而可以用来计算均方根（RMS, Root Mean Square）值
template<typename U> __device__
void cuRMSOnlineSum(
  const U curr,
  U& sigma2)
{
  sigma2 = sigma2 + curr * curr;
}

// 这段代码定义了一个名为 cuChanRMSOnlineSum 的 CUDA 设备函数，用于在线计算两个数据集平方和的累加。
// 这个函数是用于合并两个独立数据集的均方根（RMS, Root Mean Square）计算的一部分。
template<typename U> __device__
void cuChanRMSOnlineSum(
  const U sigma2B,
  U& sigma2)
{
  sigma2 = sigma2 + sigma2B;
}

// cuWelfordMuSigma2 函数是一个用于 CUDA 设备的函数，专门设计来计算张量某一维度上的均值（mu）和方差（sigma2）。
// 它采用 Welford 方法进行计算以保证数值稳定性，并可选择只计算均方根（RMS）。
// template<typename T, typename U>: 模板参数，用于不同数据类型的张量值（T）和计算过程（U）。
// const T* __restrict__ vals: 指向张量值的指针。
// const int n1, n2: 张量的维度。n1 是进行计算的维度大小，n2 是被缩减的维度大小。
// const int i1: 在 n1 维度上当前处理的特定索引。
// U& mu, sigma2: 将被计算的均值和方差的引用。
// U* buf: 指向共享内存缓冲区的指针，用于线程间通信。
// bool rms_only: 标志，指示是否只计算 RMS（true）或均值和方差（false）。
template<typename T, typename U> __device__
void cuWelfordMuSigma2(
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const int i1,
  U& mu,
  U& sigma2,
  U* buf,
  bool rms_only)
{
  // 前提条件:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  // 初始化 count, mu, 和 sigma2 为零。
  U count = U(0);
  mu= U(0);
  sigma2 = U(0);
  // 这个条件判断确保当前线程处理的 i1 索引在张量的有效范围内。
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y; // 计算一个 CUDA 块中的线程总数。
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x; // 计算当前线程在块内的唯一线性索引
    // 将 lvals 指针设置为指向当前处理的 i1 索引处张量的开始位置。
    // vals 是整个张量数据的起始指针，i1*n2 计算出当前索引在张量中的线性位置。
    const T* lvals = vals + i1*n2;
    // 初始化一个局部变量 l，用于在接下来的循环中遍历张量的元素。这里每个线程会处理多个元素，起始位置是基于线程的索引的。
    int l = 4*thrx;
    // 这个循环以步长 4*numx 遍历张量的元素，每个线程处理四个元素（如果有足够的元素）。
    for (;  l+3 < n2;  l+=4*numx) {
      // 在每次外循环的迭代中，处理四个连续的元素。
      for (int k = 0;  k < 4;  ++k) {
        // 将当前处理的元素值转换为计算使用的数据类型（U）。
        U curr = static_cast<U>(lvals[l+k]);
        // 根据 rms_only 标志调用相应的函数来更新均值和方差或仅更新平方和（用于计算 RMS）。
        if (!rms_only) {
          cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
        } else {
          cuRMSOnlineSum<U>(curr, sigma2);
        }
      }
    }
    // 这个循环处理了之前在步长为 4*numx 的循环中未处理的张量元素。每个线程独立处理它们剩余的部分。
    for (;  l < n2;  ++l) {
      U curr = static_cast<U>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum<U>(curr,mu,sigma2,count);
      } else {
       cuRMSOnlineSum<U>(curr, sigma2);
      }
    }
    // intra-warp reductions
    // 这个循环是用于在同一个 warp 内部进行 reduce 的。
    for (int l = 0;  l <= 4;  ++l) {
      // 是在 CUDA 设备上进行 warp 内部数据交换的关键部分。
      // 这行代码用于确定在一个 warp（32个线程）内，每个线程应该从哪个“lane”（即其他线程）获取数据。
      // （1<<l）这个操作在这里用于逐步增加要从中获取数据的线程的距离。例如，当 l 为 0 时，
      // 线程将从它的“邻居”线程（即下一个线程）获取数据；当 l 为 1 时，它将从两个位置之外的线程获取数据，依此类推。
      // 这个表达式计算出当前线程应该从哪个线程获取数据。随着 l 的增加，每个线程从越来越远的线程获取数据。
      // &31是因为在一个 warp 内，线程索引是循环的。也就是说，如果一个线程的索引计算结果是 32，
      // 它实际上会从索引为 0 的线程获取数据，索引为 33 的线程实际上是索引为 1 的线程，依此类推。
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      // 是一种 warp 内部的快速数据交换操作，用于从另一个线程（srcLaneB）获取数据。
      U sigma2B = WARP_SHFL(sigma2, srcLaneB);
      // 如果不是只计算 RMS（!rms_only），则使用 cuChanOnlineSum 合并两个线程的 mu、sigma2 和 count。
      // 如果只计算 RMS，则使用 cuChanRMSOnlineSum 合并 sigma2。
      if (!rms_only) {
        U muB = WARP_SHFL(mu, srcLaneB);
        U countB = WARP_SHFL(count, srcLaneB);
        cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
      } else {
        cuChanRMSOnlineSum<U>(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    // 检查是否有多个 warp。如果 blockDim.y 大于 1，则表示块中有多个 warp 需要进行reduce操作。
    if (blockDim.y > 1) {
      // 为方差和均值的reduce操作分配共享内存。ubuf 用于存储方差和均值，ibuf 用于存储计数。
      U* ubuf = (U*)buf;
      U* ibuf = (U*)(ubuf + blockDim.y);
      // 这个循环是对 warp 间的reduce操作进行分层合并。
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        // 确保只有部分线程（warp 的上半部分）将其计算的结果写入共享内存。
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          if (!rms_only) {
            ubuf[2*wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
          ubuf[2*wrt_y+1] = sigma2;
        }
        // 同步以等待共享内存存储完毕
        __syncthreads();
        // lower half merges
        // 此部分是对 warp 间数据的合并操作。
        // 确保只有部分线程（warp 的下半部分）从共享内存中读取数据并进行合并。
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          U sigma2B = ubuf[2*threadIdx.y+1];
          if (!rms_only) {
            U muB = ubuf[2*threadIdx.y];
            U countB = ibuf[threadIdx.y];
            cuChanOnlineSum<U>(muB,sigma2B,countB,mu,sigma2,count);
          } else {
            cuChanRMSOnlineSum<U>(sigma2B,sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      // 最终的结果由块内的第一个线程（threadIdx.x == 0 && threadIdx.y == 0）计算并写入共享内存。
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      // 如果不是只计算 RMS，则还需要更新均值 mu。
      if (!rms_only) {
        mu = ubuf[0];
      }
      // 计算最终的方差。
      sigma2 = ubuf[1]/U(n2);
      // don't care about final value of count, we know count == n2
    } 
    // 如果块中只有一个 warp（blockDim.y == 1），则通过 WARP_SHFL 直接在 warp 内进行数据交换和更新。
    else {
      if (!rms_only) {
        mu = WARP_SHFL(mu, 0);
      }
      sigma2 = WARP_SHFL(sigma2/U(n2), 0);
    }
  }
}

// 这个函数是上面的 cuWelfordMuSigma2 的Half特化，就不重复解析
template<> __device__
void cuWelfordMuSigma2(
  const at::Half* __restrict__ vals,
  const int n1,
  const int n2,
  const int i1,
  float& mu,
  float& sigma2,
  float* buf,
  bool rms_only)
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensor is contiguous
  // 3) 2*blockDim.y*sizeof(U)+blockDim.y*sizeof(int) shared memory available.
  //
  // compute variance and mean over n2
  float count = 0.0f;
  mu= float(0);
  sigma2 = float(0);
  if (i1 < n1) {
    // one warp normalizes one n1 index,
    // synchronization is implicit
    // initialize with standard Welford algorithm
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const at::Half* lvals = vals + i1*n2;
    int l = 8*thrx;
    if ((((size_t)lvals)&3) != 0) {
      // 16 bit alignment
      // first thread consumes first point
      if (thrx == 0) {
        float curr = static_cast<float>(lvals[0]);
        if (!rms_only) {
          cuWelfordOnlineSum(curr,mu,sigma2,count);
        } else {
          cuRMSOnlineSum(curr, sigma2);
        }

      }
      ++l;
    }
    // at this point, lvals[l] are 32 bit aligned for all threads.
    for (;  l+7 < n2;  l+=8*numx) {
      for (int k = 0;  k < 8;  k+=2) {
        float2 curr = __half22float2(*((__half2*)(lvals+l+k)));
        if (!rms_only) {
          cuWelfordOnlineSum(curr.x,mu,sigma2,count);
          cuWelfordOnlineSum(curr.y,mu,sigma2,count);
        } else {
          cuRMSOnlineSum(curr.x, sigma2);
          cuRMSOnlineSum(curr.y, sigma2);
        }
      }
    }
    for (;  l < n2;  ++l) {
      float curr = static_cast<float>(lvals[l]);
      if (!rms_only) {
        cuWelfordOnlineSum(curr,mu,sigma2,count);
      } else {
        cuRMSOnlineSum(curr, sigma2);
      }
    }
    // intra-warp reductions
    for (int l = 0;  l <= 4;  ++l) {
      int srcLaneB = (threadIdx.x+(1<<l))&31;
      float sigma2B = WARP_SHFL(sigma2, srcLaneB);
      if (!rms_only) {
        float muB = WARP_SHFL(mu, srcLaneB);
        float countB = WARP_SHFL(count, srcLaneB);
        cuChanOnlineSum(muB,sigma2B,countB,mu,sigma2,count);
      } else {
        cuChanRMSOnlineSum(sigma2B, sigma2);
      }
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float* ubuf = (float*)buf;
      float* ibuf = (float*)(ubuf + blockDim.y);
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          ubuf[2*wrt_y+1] = sigma2;
          if (!rms_only) {
            ubuf[2*wrt_y] = mu;
            ibuf[wrt_y] = count;
          }
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          float sigma2B = ubuf[2*threadIdx.y+1];
          if (!rms_only) {
            float muB = ubuf[2*threadIdx.y];
            float countB = ibuf[threadIdx.y];
            cuChanOnlineSum(muB,sigma2B,countB,mu,sigma2,count);
          } else {
            cuChanRMSOnlineSum(sigma2B, sigma2);
          }
        }
        __syncthreads();
      }
      // threadIdx.x = 0 && threadIdx.y == 0 only thread that has correct values
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        if (!rms_only) {
          ubuf[0] = mu;
        }
        ubuf[1] = sigma2;
      }
      __syncthreads();
      if (!rms_only) {
        mu = ubuf[0];
      }
      sigma2 = ubuf[1]/float(n2);
      // don't care about final value of count, we know count == n2
    } else {
      if (!rms_only) {
        mu = WARP_SHFL(mu, 0);
      }
      sigma2 = WARP_SHFL(sigma2/float(n2), 0);
    }
  }
}

template<typename U> U rsqrt(U v) {
  return U(1) / sqrt(v);
}
template<> float rsqrt(float v) {
  return rsqrtf(v);
}
template<> double rsqrt(double v) {
  return rsqrt(v);
}

namespace {
// This is the un-specialized struct.  Note that we prevent instantiation of this
// struct by putting an undefined symbol in the function body so it won't compile.
//  template <typename T>
//  struct SharedMemory
//  {
//      // Ensure that we won't compile any un-specialized types
//      __device__ T *getPointer()
//      {
//          extern __device__ void error(void);
//          error();
//          return NULL;
//      }
//  };
// https://github.com/NVIDIA/apex/issues/246
template <typename T>
struct SharedMemory;

template <>
struct SharedMemory <float>
{
    __device__ float *getPointer()
    {
        extern __shared__ float s_float[];
        return s_float;
    }
};

template <>
struct SharedMemory <double>
{
    __device__ double *getPointer()
    {
        extern __shared__ double s_double[];
        return s_double;
    }
};
}

template<typename T, typename U, typename V> __device__
void cuApplyLayerNorm_(
  V* __restrict__ output_vals,
  U* __restrict__ mean,
  U* __restrict__ invvar,
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const V* __restrict__ gamma,
  const V* __restrict__ beta,
  bool rms_only
  )
{
  // Assumptions:
  // 1) blockDim.x == warpSize
  // 2) Tensors are contiguous
  //
  for (auto i1=blockIdx.y; i1 < n1; i1 += gridDim.y) {
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    U mu,sigma2;
    cuWelfordMuSigma2(vals,n1,n2,i1,mu,sigma2,buf,rms_only);

    const T* lvals = vals + i1*n2;
    V* ovals = output_vals + i1*n2;
    U c_invvar = rsqrt(sigma2 + epsilon);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL && (beta != NULL || rms_only)) {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] = gamma[i] * static_cast<V>(c_invvar * (curr - mu)) + beta[i];
        } else {
          ovals[i] = gamma[i] * static_cast<V>(c_invvar * curr);
        }

      }
    } else {
      for (int i = thrx;  i < n2;  i+=numx) {
        U curr = static_cast<U>(lvals[i]);
        if (!rms_only) {
          ovals[i] = static_cast<V>(c_invvar * (curr - mu));
        } else {
          ovals[i] = static_cast<V>(c_invvar * curr);
        }
      }
    }
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      if (!rms_only) {
        mean[i1] = mu;
      }
      invvar[i1] = c_invvar;
    }
    __syncthreads();
  }
}

template<typename T, typename U, typename V=T> __global__
void cuApplyLayerNorm(
  V* __restrict__ output_vals,
  U* __restrict__ mean,
  U* __restrict__ invvar,
  const T* __restrict__ vals,
  const int n1,
  const int n2,
  const U epsilon,
  const V* __restrict__ gamma,
  const V* __restrict__ beta
  )
{
  cuApplyLayerNorm_<T, U, V>(output_vals, mean, invvar, vals, n1, n2, epsilon, gamma, beta, false);
}

template<typename V> __device__
V clamp_by_magnitude(V curr_gamma, double eps)
{
  const V kMinGamma = V(eps);
  if (curr_gamma >= 0) {
    if (curr_gamma < kMinGamma) {
      return kMinGamma;
    } else {
      return curr_gamma;
    }
  } else {
    if (curr_gamma > -kMinGamma) {
      return -kMinGamma;
    } else {
      return curr_gamma;
    }
  }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __device__
void cuLoadWriteStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input_or_output,
    const V* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    const double eps,
    bool rms_only
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*n2+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U c_h = static_cast<U>(input_or_output[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          warp_buf1[write_idx] = curr_dout;
          if (MemoryEfficient) {
            U curr_beta = static_cast<U>(beta[i2]);
            warp_buf2[write_idx] = curr_dout * (c_h - curr_beta) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] = curr_dout * (c_h - mean[i1]) * invvar[i1];
          }
        } else {
          if (MemoryEfficient) {
            warp_buf2[write_idx] = curr_dout * (c_h) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] = curr_dout * (c_h) * invvar[i1];
          }
        }
      } else {
        if (!rms_only) {
          warp_buf1[write_idx] = U(0);
        }
        warp_buf2[write_idx] = U(0);
      }
    }
  } else {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (!rms_only) {
        warp_buf1[write_idx] = U(0);
      }
      warp_buf2[write_idx] = U(0);
    }
  }
}

template<typename T, typename U, typename V, bool MemoryEfficient> __device__
void cuLoadAddStridedInputs(
    const int i1_block,
    const int thr_load_row_off,
    const int thr_load_col_off,
    const int i2_off,
    const int row_stride,
    U* warp_buf1,
    U* warp_buf2,
    const T* input_or_output,
    const V* dout,
    const int i1_end,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    const double eps,
    bool rms_only
    )
{
  int i1 = i1_block+thr_load_row_off;
  if (i1 < i1_end) {
    for (int k = 0;  k < blockDim.y;  ++k) {
      int i2 = i2_off + k;
      int load_idx = i1*n2+i2;
      int write_idx = thr_load_row_off*row_stride+thr_load_col_off+k;
      if (i2<n2) {
        U c_h = static_cast<U>(input_or_output[load_idx]);
        U curr_dout = static_cast<U>(dout[load_idx]);
        if (!rms_only) {
          U curr_beta = static_cast<U>(beta[i2]);
          warp_buf1[write_idx] += curr_dout;
          if (MemoryEfficient) {
            warp_buf2[write_idx] += curr_dout * (c_h - curr_beta) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] += curr_dout * (c_h - mean[i1]) * invvar[i1];
          }
        } else {
          if (MemoryEfficient) {
            warp_buf2[write_idx] += curr_dout * (c_h) / static_cast<U>(clamp_by_magnitude(gamma[i2], eps));
          } else {
            warp_buf2[write_idx] += curr_dout * (c_h) * invvar[i1];
          }
        }
      }
    }
  }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __global__
void cuComputePartGradGammaBeta(
    const V* __restrict__ dout,
    const T* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    const V* __restrict__ gamma,
    const V* __restrict__ beta,
    U* part_grad_gamma,
    U* part_grad_beta,
    const double eps,
    bool rms_only)
{
    const int numsegs_n1 = (n1+blockDim.y*blockDim.y-1) / (blockDim.y*blockDim.y);
    const int segs_per_block = (numsegs_n1 + gridDim.y - 1) / gridDim.y;
    const int i1_beg = blockIdx.y * segs_per_block * blockDim.y*blockDim.y;
    const int i1_beg_plus_one = (blockIdx.y+1) * segs_per_block * blockDim.y*blockDim.y;
    const int i1_end = i1_beg_plus_one < n1 ? i1_beg_plus_one : n1;
    const int row_stride = blockDim.x+1;
    const int thr_load_col_off = (threadIdx.x*blockDim.y)&(blockDim.x-1);
    const int thr_load_row_off = (threadIdx.x*blockDim.y)/blockDim.x + threadIdx.y*blockDim.y;
    const int i2_off = blockIdx.x * blockDim.x + thr_load_col_off;
    SharedMemory<U> shared;
    U* buf = shared.getPointer(); // buf has at least blockDim.x * blockDim.y * blockDim.y + (blockDim.y - 1)*(blockDim.x/blockDim.y) elements
    U* warp_buf1 = (U*)buf;
    U* warp_buf2 = warp_buf1 + blockDim.y * blockDim.y * row_stride;
    // compute partial sums from strided inputs
    // do this to increase number of loads in flight
    cuLoadWriteStridedInputs<T, U, V, MemoryEfficient>(i1_beg,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input_or_output,dout,i1_end,n2,mean,invvar,gamma,beta,eps, rms_only);
    for (int i1_block = i1_beg+blockDim.y*blockDim.y;  i1_block < i1_end;  i1_block+=blockDim.y*blockDim.y) {
      cuLoadAddStridedInputs<T, U, V, MemoryEfficient>(i1_block,thr_load_row_off,thr_load_col_off,i2_off,row_stride,warp_buf1,warp_buf2,input_or_output,dout,i1_end,n2,mean,invvar,gamma,beta,eps, rms_only);
    }
    __syncthreads();
    // inter-warp reductions
    // sum within each warp
    U acc1 = U(0);
    U acc2 = U(0);
    for (int k = 0;  k < blockDim.y;  ++k) {
      int row1 = threadIdx.y + k*blockDim.y;
      int idx1 = row1*row_stride + threadIdx.x;
      if (!rms_only) {
        acc1 += warp_buf1[idx1];
      }
      acc2 += warp_buf2[idx1];
    }
    if (!rms_only) {
      warp_buf1[threadIdx.y*row_stride+threadIdx.x] = acc1;
    }
    warp_buf2[threadIdx.y*row_stride+threadIdx.x] = acc2;
    __syncthreads();
    // sum all warps
    for (int offset = blockDim.y/2;  offset > 1;  offset /= 2) {
      if (threadIdx.y < offset) {
        int row1 = threadIdx.y;
        int row2 = threadIdx.y + offset;
        int idx1 = row1*row_stride + threadIdx.x;
        int idx2 = row2*row_stride + threadIdx.x;
        if (!rms_only) {
          warp_buf1[idx1] += warp_buf1[idx2];
        }
        warp_buf2[idx1] += warp_buf2[idx2];
      }
      __syncthreads();
    }
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadIdx.y == 0 && i2 < n2) {
      int row1 = threadIdx.y;
      int row2 = threadIdx.y + 1;
      int idx1 = row1*row_stride + threadIdx.x;
      int idx2 = row2*row_stride + threadIdx.x;
      if (!rms_only) {
        part_grad_beta[blockIdx.y*n2+i2] = warp_buf1[idx1] + warp_buf1[idx2];
      }
      part_grad_gamma[blockIdx.y*n2+i2] = warp_buf2[idx1] + warp_buf2[idx2];
    }
}

template<typename U, typename V> __global__
void cuComputeGradGammaBeta(
    const U* part_grad_gamma,
    const U* part_grad_beta,
    const int part_size,
    const int n1,
    const int n2,
    V* grad_gamma,
    V* grad_beta,
    bool rms_only)
{
    // sum partial gradients for gamma and beta
    SharedMemory<U> shared;
    U* buf = shared.getPointer();
    int i2 = blockIdx.x * blockDim.x + threadIdx.x;
    if (i2 < n2) {
      // each warp does sequential reductions until reduced part_size is num_warps
      int num_warp_reductions = part_size / blockDim.y;
      U sum_gamma = U(0);
      U sum_beta = U(0);
      const U* part_grad_gamma_ptr = part_grad_gamma + threadIdx.y * num_warp_reductions * n2 + i2;
      const U* part_grad_beta_ptr = part_grad_beta + threadIdx.y * num_warp_reductions * n2 + i2;
      for (int warp_offset = 0;  warp_offset < num_warp_reductions;  ++warp_offset) {
        sum_gamma += part_grad_gamma_ptr[warp_offset*n2];
        if (!rms_only) {
          sum_beta += part_grad_beta_ptr[warp_offset*n2];
        }
      }
      // inter-warp reductions
      const int nbsize3 = blockDim.x * blockDim.y / 2;
      for (int offset = blockDim.y/2;  offset >= 1;  offset /= 2) {
        // top half write to shared memory
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int write_idx = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          buf[write_idx] = sum_gamma;
          if (!rms_only) {
            buf[write_idx+nbsize3] = sum_beta;
          }
        }
        __syncthreads();
        // bottom half sums
        if (threadIdx.y < offset) {
          const int read_idx = threadIdx.y * blockDim.x + threadIdx.x;
          sum_gamma += buf[read_idx];
          if (!rms_only) {
            sum_beta += buf[read_idx+nbsize3];
          }
        }
        __syncthreads();
      }
      // write out fully summed gradients
      if (threadIdx.y == 0) {
        grad_gamma[i2] = sum_gamma;
        if (!rms_only) {
          grad_beta[i2] = sum_beta;
        }
      }
    }
}


template<typename T, typename U, typename V, bool MemoryEfficient> __global__
void cuComputeGradInput(
    const V* __restrict__ dout,
    const T* __restrict__ input_or_output,
    const int n1,
    const int n2,
    const U* __restrict__ mean,
    const U* __restrict__ invvar,
    U epsilon,
    const V* gamma,
    const V* beta,
    T* grad_input,
    const double eps,
    bool rms_only)
{
  for (auto i1=blockIdx.y; i1 < n1; i1 += gridDim.y) {
    U sum_loss1 = U(0);
    U sum_loss2 = U(0);
    const T* k_h = input_or_output + i1*n2;
    const V* k_dout = dout + i1*n2;
    const U c_invvar = invvar[i1];
    const U c_mean = !MemoryEfficient ? mean[i1] : 0.;
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    if (gamma != NULL) {
      int l = 4*thrx;
      for (;  l+3 < n2;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_h[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          if (!rms_only) {
            sum_loss1 += c_loss * gamma[l+k];
            if (MemoryEfficient) {
              sum_loss2 += c_loss * (c_h - beta[l+k]);
            } else {
              sum_loss2 += c_loss * gamma[l+k] * (c_h - c_mean) * c_invvar;
            }
          } else {
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * gamma[l+k] * (c_h) * c_invvar;
            }
          }
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss * gamma[l];
          if (MemoryEfficient) {
            sum_loss2 += c_loss * (c_h - beta[l]);
          } else {
            sum_loss2 += c_loss * gamma[l] * (c_h - c_mean) * c_invvar;
          }
        } else {
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * gamma[l] * (c_h) * c_invvar;
          }
        }
      }
    } else {
      int l = 4*thrx;
      for (;  l+3 < n2;  l+=4*numx) {
        for (int k = 0;  k < 4;  ++k) {
          const U c_h = static_cast<U>(k_h[l+k]);
          const U c_loss = static_cast<U>(k_dout[l+k]);
          if (!rms_only) {
            sum_loss1 += c_loss;
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
            }
          } else {
            if (MemoryEfficient) {
              sum_loss2 += c_loss * c_h;
            } else {
              sum_loss2 += c_loss * (c_h) * c_invvar;
            }
          }
        }
      }
      for (;  l < n2;  ++l) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        if (!rms_only) {
          sum_loss1 += c_loss;
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * (c_h - c_mean) * c_invvar;
          }
        } else {
          if (MemoryEfficient) {
            sum_loss2 += c_loss * c_h;
          } else {
            sum_loss2 += c_loss * (c_h) * c_invvar;
          }
        }
      }
    }
    // intra-warp reductions
    for (int mask = blockDim.x/2;  mask > 0;  mask /= 2) {
      if (!rms_only) {
        sum_loss1 += WARP_SHFL_XOR(sum_loss1, mask);
      }
      sum_loss2 += WARP_SHFL_XOR(sum_loss2, mask);
    }
    // inter-warp reductions
    if (blockDim.y > 1) {
      SharedMemory<U> shared;
      U* buf = shared.getPointer();
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_i = (threadIdx.y - offset) * blockDim.x + threadIdx.x;
          if (!rms_only) {
            buf[2*wrt_i] = sum_loss1;
          }
          buf[2*wrt_i+1] = sum_loss2;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.y < offset) {
          const int read_i = threadIdx.y * blockDim.x + threadIdx.x;
          if (!rms_only) {
            sum_loss1 += buf[2*read_i];
          }
          sum_loss2 += buf[2*read_i+1];
        }
        __syncthreads();
      }
      if (threadIdx.y == 0) {
        if (!rms_only) {
          buf[2*threadIdx.x] = sum_loss1;
        }
        buf[2*threadIdx.x+1] = sum_loss2;
      }
      __syncthreads();
      if (threadIdx.y !=0) {
        if (!rms_only) {
          sum_loss1 = buf[2*threadIdx.x];
        }
        sum_loss2 = buf[2*threadIdx.x+1];
      }
    }
    // all threads now have the two sums over l
    U fH = (U)n2;
    U term1 = (U(1) / fH) * c_invvar;
    T* k_grad_input = grad_input + i1*n2;
    if (gamma != NULL) {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        const U k_gamma = static_cast<U>(clamp_by_magnitude(gamma[l], eps));
        U f_grad_input = fH * c_loss * k_gamma;
        if (!rms_only) {
          const U k_beta = beta[l];
          f_grad_input -= sum_loss1;
          if (MemoryEfficient) {
            f_grad_input -= (c_h - k_beta) / k_gamma * sum_loss2;
          } else {
            f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
          }
        } else {
          if (MemoryEfficient) {
            f_grad_input -= c_h / k_gamma * sum_loss2;
          } else {
            f_grad_input -= c_h * c_invvar * sum_loss2;
          }
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    } else {
      for (int l = thrx;  l < n2;  l+=numx) {
        const U c_h = static_cast<U>(k_h[l]);
        const U c_loss = static_cast<U>(k_dout[l]);
        U f_grad_input = fH * c_loss;
        if (!rms_only) {
          f_grad_input -= sum_loss1;
          if (MemoryEfficient) {
            f_grad_input -= c_h * sum_loss2;
          } else {
            f_grad_input -= (c_h - c_mean) * c_invvar * sum_loss2;
          }
        } else {
          if (MemoryEfficient) {
            f_grad_input -= c_h * sum_loss2;
          } else {
            f_grad_input -= c_h * c_invvar * sum_loss2;
          }
        }
        f_grad_input *= term1;
        k_grad_input[l] = static_cast<T>(f_grad_input);
      }
    }
    // prevent race where buf is written again before reads are done
    __syncthreads();
  }
}


template<typename T, typename U, typename V=T>
void HostApplyLayerNorm(
    V* output,
    U* mean,
    U* invvar,
    const T* input,
    int n1,
    int n2,
    double epsilon,
    const V* gamma,
    const V* beta
    )
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    const dim3 threads(32,4,1);
    const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
    const dim3 blocks(1, std::min((uint64_t)n1, maxGridY), 1);
    int nshared =
        threads.y > 1 ?
            threads.y*sizeof(U)+(threads.y/2)*sizeof(U) :
            0;
    cuApplyLayerNorm<<<blocks, threads, nshared, stream>>>(
      output, mean, invvar, input, n1, n2, U(epsilon), gamma, beta);
}

void cuda_layer_norm(
    at::Tensor* output,
    at::Tensor* mean,
    at::Tensor* invvar,
    at::Tensor* input,
    int n1,
    int n2,
    #ifdef VERSION_GE_1_1
    at::IntArrayRef normalized_shape,
    #else
    at::IntList normalized_shape,
    #endif
    at::Tensor* gamma,
    at::Tensor* beta,
    double epsilon)
{
    using namespace at;
    DISPATCH_DOUBLE_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
        input->scalar_type(), output->scalar_type(), "layer_norm_cuda_kernel",
        using accscalar_t = at::acc_type<scalar_t_in, true>;
        HostApplyLayerNorm<scalar_t_in, accscalar_t, scalar_t_out>(
          output->DATA_PTR<scalar_t_out>(),
              mean->DATA_PTR<accscalar_t>(),
          invvar->DATA_PTR<accscalar_t>(),
          input->DATA_PTR<scalar_t_in>(),
          n1,n2,
          epsilon,
          gamma != NULL ? gamma->DATA_PTR<scalar_t_out>() : NULL,
          beta != NULL ? beta->DATA_PTR<scalar_t_out>() : NULL);
      )
}


template<typename T, typename U=float, typename V=T>
void HostLayerNormGradient(
    const V* dout,
    const U* mean,
    const U* invvar,
    at::Tensor* input_or_output,
    int n1,
    int n2,
    const V* gamma,
    const V* beta,
    double epsilon,
    T* grad_input,
    V* grad_gamma,
    V* grad_beta,
    bool memory_efficient
    )
{
    auto stream = at::cuda::getCurrentCUDAStream().stream();

    if (gamma != NULL && beta != NULL) {
      // compute grad_gamma(j) and grad_beta(j)
      const int part_size = 16;
      const dim3 threads2(32,4,1);
      const dim3 blocks2((n2+threads2.x-1)/threads2.x,part_size,1);
      const int nshared2_a = 2 * sizeof(U) * threads2.y * threads2.y * (threads2.x + 1);
      const int nshared2_b = threads2.x * threads2.y * sizeof(U);
      const int nshared2 = nshared2_a > nshared2_b ? nshared2_a : nshared2_b;
      // note (mkozuki): I can hard code part_grad_gamma's dtype as float given that
      // the `cuda_layer_norm_gradient` doesn't support double.
      const auto part_grad_dtype =
        (input_or_output->scalar_type() == at::ScalarType::Half || input_or_output->scalar_type() == at::ScalarType::BFloat16) ?
        at::ScalarType::Float :
        input_or_output->scalar_type();
      at::Tensor part_grad_gamma = at::empty({part_size,n2}, input_or_output->options().dtype(part_grad_dtype));
      at::Tensor part_grad_beta = at::empty_like(part_grad_gamma);
      BOOL_SWITCH(memory_efficient, MemoryEfficient, [&]{
        auto kernel = &cuComputePartGradGammaBeta<T, U, V, MemoryEfficient>;
        kernel<<<blocks2, threads2, nshared2, stream>>>(
                        dout,
                        input_or_output->DATA_PTR<T>(),
                        n1,n2,
                        mean,
                        invvar,
                        U(epsilon),
                        gamma,
                        beta,
                        part_grad_gamma.DATA_PTR<U>(),
                        part_grad_beta.DATA_PTR<U>(),
                        epsilon,
                        false);
      });

      const dim3 threads3(32,8,1);
      const dim3 blocks3((n2+threads2.x-1)/threads2.x,1,1);
      const int nshared3 = threads3.x * threads3.y * sizeof(U);
      cuComputeGradGammaBeta<<<blocks3, threads3, nshared3, stream>>>(
                      part_grad_gamma.DATA_PTR<U>(),
                      part_grad_beta.DATA_PTR<U>(),
                      part_size,
                      n1,n2,
                      grad_gamma,
                      grad_beta,
                      false);
    }

    // compute grad_input
    const uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
    const dim3 blocks1(1, std::min((uint64_t)n1, maxGridY), 1);
    const dim3 threads1(32,4,1);
    int nshared =
            threads1.y > 1 ?
            threads1.y*threads1.x*sizeof(U) :
            0;
    BOOL_SWITCH(memory_efficient, MemoryEfficient, [&] {
      auto kernel = cuComputeGradInput<T, U, V, MemoryEfficient>;
      kernel<<<blocks1, threads1, nshared, stream>>>(
              dout,
              input_or_output->DATA_PTR<T>(),
              n1,n2,
              mean,
              invvar,
              U(epsilon),
              gamma,
              beta,
              grad_input,
              epsilon,
              false);
    });
}

void cuda_layer_norm_gradient(
    at::Tensor* dout,
    at::Tensor* mean,
    at::Tensor* invvar,
    at::Tensor* input_or_output,
    int n1,
    int n2,
    #ifdef VERSION_GE_1_1
    at::IntArrayRef normalized_shape,
    #else
    at::IntList normalized_shape,
    #endif
    at::Tensor* gamma,
    at::Tensor* beta,
    double epsilon,
    at::Tensor* grad_input,
    at::Tensor* grad_gamma,
    at::Tensor* grad_beta,
    bool memory_efficient)
{
    using namespace at;
    // we can do away with `accscalar_t` as there're only three dtypes: fp32, fp16, bf16
    DISPATCH_FLOAT_HALF_AND_BFLOAT_INOUT_TYPES(
      input_or_output->scalar_type(), gamma == NULL ? input_or_output->scalar_type() :  gamma->scalar_type(), "cuComputeGradInput",
      using accscalar_t = at::acc_type<scalar_t_in, true>;
      HostLayerNormGradient(
        dout->DATA_PTR<scalar_t_out>(),
        mean != NULL ? mean->DATA_PTR<accscalar_t>() : NULL,
        invvar->DATA_PTR<accscalar_t>(),
        input_or_output,
        n1,n2,
            // TMJ pass NULL argument for gamma, beta, grad_gamma and grad_beta
            // if gamma Tensor is NULL on input.
        gamma != NULL ? gamma->DATA_PTR<scalar_t_out>() : NULL,
        gamma != NULL ? beta->DATA_PTR<scalar_t_out>() : NULL,
        epsilon,
        grad_input->DATA_PTR<scalar_t_in>(),
        gamma != NULL ? grad_gamma->DATA_PTR<scalar_t_out>() : NULL,
        gamma != NULL ? grad_beta->DATA_PTR<scalar_t_out>() : NULL,
        memory_efficient);
    )
}
